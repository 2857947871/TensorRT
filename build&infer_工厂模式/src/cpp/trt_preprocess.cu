#include "hip/hip_runtime.h"
# include <stdio.h>
# include <iostream>
# include "hip/hip_runtime_api.h"
# include "trt_preprocess.hpp"

namespace process {

__global__ void nearest_BGR2RGB_nhwc2nchw_norm_kernel(
    float* tar, const uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH,
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{
    // nearest neighbour -- resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // nearest neighbour -- 计算最近坐标
    int src_y = floor((float)y * scaled_h);
    int src_x = floor((float)x * scaled_w);

    if (src_x < 0 || src_y < 0 || src_x > srcW || src_y > srcH) {
        // nearest neighbour -- 对于越界的部分，不进行计算
    } else {
        // nearest neighbour -- 计算tar中对应坐标的索引
        int tarIdx  = y * tarW + x;
        int tarArea = tarW * tarH;

        // nearest neighbour -- 计算src中最近邻坐标的索引
        int srcIdx = (src_y * srcW + src_x) * 3;

        // nearest neighbour -- 实现nearest beighbour的resize + BGR2RGB + nhwc2nchw + norm
        tar[tarIdx + tarArea * 0] = (src[srcIdx + 2] / 255.0f - d_mean[2]) / d_std[2];
        tar[tarIdx + tarArea * 1] = (src[srcIdx + 1] / 255.0f - d_mean[1]) / d_std[1];
        tar[tarIdx + tarArea * 2] = (src[srcIdx + 0] / 255.0f - d_mean[0]) / d_std[0];
    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_norm_kernel(
    float* tar, const uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{

    // bilinear interpolation -- resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = ((y + 0.5) * scaled_h - 0.5) - src_y1;
        float tw   = ((x + 0.5) * scaled_w - 0.5) - src_x1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        float a1_1 = (1.0 - tw) * (1.0 - th);  //右下
        float a1_2 = tw * (1.0 - th);          //左下
        float a2_1 = (1.0 - tw) * th;          //右上
        float a2_2 = tw * th;                  //左上

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  //左上
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  //右上
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  //左下
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  //右下

        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = y * tarW  + x;
        int tarArea   = tarW * tarH;

        // bilinear interpolation -- 实现bilinear interpolation的resize + BGR2RGB + NHWC2NCHW normalization
        // 注意，这里tar和src进行遍历的方式是不一样的
        tar[tarIdx + tarArea * 0] = 
            (round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f - d_mean[2]) / d_std[2];

        tar[tarIdx + tarArea * 1] = 
            (round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f - d_mean[1]) / d_std[1];

        tar[tarIdx + tarArea * 2] = 
            (round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f - d_mean[0]) / d_std[0];

    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_shift_norm_kernel(
    float* tar, const uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{
    // resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = (float)y * scaled_h - src_y1;
        float tw   = (float)x * scaled_w - src_x1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        float a1_1 = (1.0 - tw) * (1.0 - th);  // 右下
        float a1_2 = tw * (1.0 - th);          // 左下
        float a2_1 = (1.0 - tw) * th;          // 右上
        float a2_2 = tw * th;                  // 左上

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  // 左上
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  // 右上
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  // 左下
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  // 右下

        // bilinear interpolation -- 计算原图在目标图中的x, y方向上的偏移量
        y = y - int(srcH / (scaled_h * 2)) + int(tarH / 2);
        x = x - int(srcW / (scaled_w * 2)) + int(tarW / 2);

        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = (y * tarW  + x) * 3;
        int tarArea   = tarW * tarH;

        // bilinear interpolation -- 实现bilinear interpolation + BGR2RGB + shift + nhwc2nchw
        tar[tarIdx + tarArea * 0] = 
            (round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f - d_mean[2]) / d_std[2];

        tar[tarIdx + tarArea * 1] = 
            (round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f - d_mean[1]) / d_std[1];

        tar[tarIdx + tarArea * 2] = 
            (round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f - d_mean[0]) / d_std[0];
    }
}

void resize_gpu(const uint8_t* d_src, float* d_tar, 
    int srcW, int srcH, int tarW, int tarH, 
    float* d_mean, float* d_std, process::tactics tac)
{
    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(tarW / 32 + 1, tarH / 32 + 1, 1);
   
    //scaled resize
    float scaled_h = (float)srcH / tarH;
    float scaled_w = (float)srcW / tarW;
    float scale = (scaled_h > scaled_w ? scaled_h : scaled_w);

    switch (tac) {
    case process::tactics::GPU_NEAREST:
        nearest_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h, d_mean, d_std);
        break;
    case process::tactics::GPU_NEAREST_CENTER:
        nearest_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale, d_mean, d_std);
        break;
    case process::tactics::GPU_BILINEAR:
        bilinear_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h, d_mean, d_std);
        break;
    case process::tactics::GPU_BILINEAR_CENTER:
        bilinear_BGR2RGB_nhwc2nchw_shift_norm_kernel 
                <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale, d_mean, d_std);
        break;
    default:
        LOGE("ERROR: Wrong GPU resize tactics selected. Program terminated");
        exit(1);
    }
}
}; // namespace process, 与.hpp对齐